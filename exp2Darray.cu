
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define ROWS 4
#define COLUMNS 5

typedef struct mystruct
{
	int a[ROWS];
	int **data;
}mystruct;

__global__ void printKernel(mystruct *d_var)
{
	int i, j;
	for(i = 0; i < ROWS; i++)
	{
		for(j = 0; j < COLUMNS; j++)
		{
			printf("%d\t", d_var->data[i][j]);
		}
		printf("\n");
	}
}

int main()
{
	int i, j, k=1;
	mystruct *var, *d_var;

/* Allocate and initialize a dynamic 2D array on CPU */
	var->data = (int**)malloc(ROWS*sizeof(int*));
	for (i = 0; i < ROWS; i++)
		var->data[i] = (int*)malloc(COLUMNS*sizeof(int));

	for(i = 0; i < ROWS; i++)
	{
		var->a[i] = 2;
		for(j = 0; j < COLUMNS; j++)
		{
			var->data[i][j] = k++;
		}		
	}

/* Allocate memory for struct on GPU*/
	hipMalloc((void**)&d_var, sizeof(mystruct));	
/*Allocate memory explicitly for the 2D array*/
	hipMalloc((void**)&d_var->data, ROWS*sizeof(int*));
	for(i = 0; i < ROWS; i++)
		hipMalloc((void**)&d_var->data[i], COLUMNS*sizeof(int));
/*Copy the host struct to device*/
	hipMemcpy(d_var, var, (sizeof(mystruct)+ROWS*COLUMNS*sizeof(int)), hipMemcpyHostToDevice);
	printKernel<<<1,1>>>(d_var);
	free(var);
	hipFree(d_var);
	return 0;
}
